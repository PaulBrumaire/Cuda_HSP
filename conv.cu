#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include <sys/time.h>

#define N 32
#define P 32
#define Q 6
#define K 5


void MatrixInit(float *M, int n, int p,int q){
    int i;
    for (i=0;i<n*p*q;i++){
        M[i]= (float)rand()/(RAND_MAX);
    }
}

void MatrixInit2(float *M, int n, int p,int q){
    int i;
    for (i=0;i<n*p*q;i++){
        M[i]= (float)rand()/(RAND_MAX/2)-1;
    }
}

void MatrixInit0(float *M, int n, int p,int q){
    int i;
    for (i=0;i<n*p*q;i++){
        M[i]= (float) 0;
    }
}


void MatrixInitId(float *M, int n, int p,int q){
    int i;
    for (i=0;i<n*p*q;i++){
        M[i]= (float) 0;
    }
    M[(n*p)/2]= (float) 1;
}

void MatrixPrint(float *M, int n, int p, int q){
    int i;
    for (i=0;i<n*p*q;i++){
        if(M[i]>0) printf(" ");
        printf("%1.2f ", M[i]);
        if ((i+1+p)%p==0 ){
            printf("\n");
        }
        if ((i+1)%(n*p)==0 ){
            printf("\n\n");
        }
    }
    printf("\n");
}

__device__ float cudaActivationTanh(float val) {
    // float temp = exp(2*val);
    // printf("%f\n",temp);
    // return (float) (temp-1)/(temp+1);
    return tanhf(val);
}

__global__ void cudaConv2D(float *img, float *kernels, float * out, int n, int p, int q, int k ) {
    // n,p= lignes,col img
    // q = nb kernels
    // k = dim kernel (k*k)

    int m=n-k+1; //dim img out
    //int id = (k-1)/2; //center

    int l = threadIdx.x; //->28 (dim img out)
    int c = threadIdx.y;  //->28 (dim img out)
    
    int d = blockIdx.x;  //->6  (dim nb kernel deconv)
    
    float temp=0;
    int i,j;
    //Calcul du bloc K*K
    for (int ki= 0; ki<k;ki++){
        for (int kj= 0; kj<k;kj++){
            i=l+ki;
            j=c+kj;   
            temp+=img[i*n + j] * kernels[d*k*k + ki*k + kj];
        }
    }

    out[d*m*m + l*m + c] = cudaActivationTanh(temp);//temp;//
}





__global__ void cudaConv3D(float *img, float *kernels, float * out, int n, int p, int q, int k ) {
    // n,p= lignes,col img
    // q = nb kernels
    // k = dim kernel (k*k)

    int m=n-k+1; //dim img out
    //int id = (k-1)/2; //center

    int l = threadIdx.x; //(dim img out)
    int c = threadIdx.y;  //(dim img out)
    
    int d = blockIdx.x;  //(dim nb kernel deconv)
    //int o = blockIdx.y; //dim nb images in

    // printf("/ %d",d);
    
    
    int i,j;
    for (int o=0; o<1; o++){
        //Calcul du bloc K*K =5*5
        float temp=0;
        for (int ki= 0; ki<k;ki++){
            for (int kj= 0; kj<k;kj++){
                i=l+ki;
                j=c+kj;   
                temp+=img[o*n*p +i*n + j] * kernels[d*k*k + ki*k + kj];
            }
        }

        //printf("/%d",o);//*m*m*16 +d*m*m+ l*m + c) ;
        // o=0-5, d=0-15, l=0-9, c=0-9
        out[ o*m*m*q +d*m*m+ l*m + c]= cudaActivationTanh(temp);
    }
}

__global__ void cudaCombine(float *in, float * out, float * id ) {
    int l = threadIdx.x; // (ligne)
    int c = threadIdx.y;  //(colonne)
    
    int g = blockIdx.x; // (dim out profondeur)

    float temp = 0;
    for(int i=0;i<6;i++){
        temp+=in[i*16*10*10 + g*10*10 +l*10 +c]*id[i*16+g];
    }
    out[g*10*10 + l*10 + c ]= temp;
}

__global__ void cudaMeanPool(float *in, float *out, int n, int p, int q) {
    // n,p= lignes,col in
    // q = nb kernels = profondeur

    int m=n/2; //dim out

    int l = threadIdx.x; //->28 (dim out)
    int c = threadIdx.y;  //->28 (dim out)
    int d = blockIdx.x;  //->6  (dim nb kernel)
    
    float temp=0;
    int i,j;
    //Calcul du bloc K*K
    for (int ki= 0; ki<2;ki++){
        for (int kj= 0; kj<2;kj++){
            i=l+ki;
            j=c+kj;    
            temp+=in[d*n*p + i*n + j];
        }
    }

    out[d*m*m + l*m + c]= temp/4;
}




int main(){
    int L=(N-K+1); //dim out conv
    int M=(L/2); //dim out pool

    float *raw_data, *C1_data, *C1_kernel, *S2_data,*C3_dataTemp, *C3_data , *C3_kernel;

    srand(time(NULL));

    raw_data = (float*)malloc(N*P * sizeof(float));
    C1_data = (float*)malloc(Q*L*L * sizeof(float));
    C1_kernel = (float*)malloc(Q*K*K * sizeof(float));
    S2_data = (float*)malloc(Q*M*M * sizeof(float));

    C3_dataTemp = (float*)malloc(96*10*10 * sizeof(float));
    C3_data = (float*)malloc(16*10*10 * sizeof(float));
    C3_kernel = (float*)malloc(16*K*K * sizeof(float));

    float combineId[96] = {
        1,0,0,0,1,1,1,0,0,1,1,1,1,0,1,1,
        1,1,0,0,0,1,1,1,0,0,1,1,1,1,0,1,
        1,1,1,0,0,0,1,1,1,0,0,1,0,1,1,1,
        0,1,1,1,0,0,1,1,1,1,0,0,1,0,1,1,
        0,0,1,1,1,0,0,1,1,1,1,0,1,1,0,1,
        0,0,0,1,1,1,0,0,1,1,1,1,0,1,1,1
    };

    MatrixInit(raw_data,N,P,1);
    MatrixInit0(C1_data,L,L,Q);
    MatrixInit2(C1_kernel,K,K,Q);

    MatrixInit0(S2_data,M,M,Q);

    MatrixInit0(C3_dataTemp,10,10,96);
    MatrixInit0(C3_data,10,10,16);
    MatrixInit2(C3_kernel,K,K,16);

    //MatrixPrint(combineId,6,16,1);


    printf("IMAGE\n");
    MatrixPrint(raw_data,N,P,1);
    printf("KERNEL\n");
    //MatrixPrint(C1_kernel,K,K,1);
    //GPU
    float *d_combine,*d_raw, *d_C1, *d_C1_kernel, *d_S2,*d_C3Temp, *d_C3, *d_C3_kernel;

    //CUDA ARRRAY---------------------------------------------------------------------
    hipMalloc((void**)&d_raw, sizeof(float)*N*P);
    hipMalloc((void**)&d_C1, sizeof(float)*Q*L*L);
    hipMalloc((void**)&d_C1_kernel, sizeof(float)*Q*K*K);
    hipMalloc((void**)&d_S2, sizeof(float)*Q*M*M);
    hipMalloc((void**)&d_C3Temp, sizeof(float)*96*10*10);
    hipMalloc((void**)&d_C3, sizeof(float)*16*10*10);
    hipMalloc((void**)&d_C3_kernel, sizeof(float)*16*K*K);
    hipMalloc((void**)&d_combine, sizeof(float)*16*6);

    hipMemcpy(d_raw, raw_data, sizeof(float) * N*P, hipMemcpyHostToDevice);
    hipMemcpy(d_C1, C1_data, sizeof(float) * Q*L*L, hipMemcpyHostToDevice);
    hipMemcpy(d_C1_kernel, C1_kernel, sizeof(float) * Q*K*K, hipMemcpyHostToDevice);
    hipMemcpy(d_S2, S2_data, sizeof(float) * Q*M*M, hipMemcpyHostToDevice);
    hipMemcpy(d_C3Temp, C3_dataTemp, sizeof(float) * 96*10*10, hipMemcpyHostToDevice);
    hipMemcpy(d_C3, C3_data, sizeof(float) * 16*10*10, hipMemcpyHostToDevice);
    hipMemcpy(d_C3_kernel, C3_kernel, sizeof(float) * 16*K*K, hipMemcpyHostToDevice);
    hipMemcpy(d_combine, combineId, sizeof(float) * 16*6, hipMemcpyHostToDevice);


    //CONV 1---------------------------------------------------------------------
    dim3 nb_thread(L,L);//L=28
    dim3 nb_block(Q);//Q=6

    cudaConv2D<<<nb_block,nb_thread>>>(d_raw, d_C1_kernel, d_C1, N, P, Q, K );
    hipDeviceSynchronize();

    //MEAN 1---------------------------------------------------------------------
    dim3 nb_thread2(M,M);//M=14
    dim3 nb_block2(Q);//Q=6
    cudaMeanPool<<<nb_block2,nb_thread2>>>(d_C1, d_S2, L, L, Q );
    hipDeviceSynchronize();

    //CONV 2  14*14*6 -> 10*10*96 -> 10*10*16 ---------------------------------------------------------------------
    dim3 nb_thread3(10,10); //im out
    dim3 nb_block3(16); //nb kernel
    cudaConv3D<<<nb_block3,nb_thread3>>>(d_S2, d_C3_kernel, d_C3Temp, 14, 14, 16, K );


    //COMBINE ---------------------------------------------------------------------------------
    hipDeviceSynchronize();
    dim3 nb_thread4(10,10);
    dim3 nb_block4(16);
    cudaCombine<<<nb_block4,nb_thread4>>>(d_C3Temp,d_C3, d_combine);
    hipDeviceSynchronize();


    //ARRAY COPY TO CPU --------------------------------------------------------------------------
    hipMemcpy(C1_data, d_C1, sizeof(float)* Q*L*L, hipMemcpyDeviceToHost);
    printf("OUT\n");
    MatrixPrint(C1_data,L,L,1);
    
    hipMemcpy(S2_data, d_S2, sizeof(float)* Q*M*M, hipMemcpyDeviceToHost);
    printf("MEAN\n");
    //MatrixPrint(S2_data,M,M,1); //M<L

    hipMemcpy(C3_dataTemp, d_C3Temp, sizeof(float)* 96*10*10, hipMemcpyDeviceToHost);
    hipMemcpy(C3_data, d_C3, sizeof(float)* 16*10*10, hipMemcpyDeviceToHost);
    
    printf("C3 Temp\n");
    //MatrixPrint(C3_dataTemp,10,10,16); 
    printf("C3\n");
    MatrixPrint(C3_data,10,10,16); //M<L
    
    hipDeviceSynchronize();
    
    hipFree(d_raw);
    hipFree(d_C1);
    hipFree(d_C1_kernel);

    free(raw_data);
    free(C1_data);
    free(C1_kernel);
}