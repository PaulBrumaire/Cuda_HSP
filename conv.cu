#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include <sys/time.h>

#define N 32
#define P 32
#define Q 6
#define K 5
#define WIDTH 28
#define HEIGHT 28

void vector_add(float *in) {
    float temp =0;
    for(int i = 0; i < 10; i++){
        temp += in[i];
    }
    printf("%f \n",temp);
}

void indexMax(float *in) {
    int k = 0;
    float max = in[k];

    for (int i = 0; i < 10; ++i)
    {
        if (in[i] > max)
        {
            max = (float)in[i];
            k = i;
        }
    }
    printf("C'est %d \n",k);
}

void read_file(char* path, float * out){
    FILE *f = fopen(path, "r");

    if (f == NULL)
    {
        printf("Error: could not open file %s", path);
    }
    int i =0;

    while ((fscanf(f,"%f", &out[i])) != EOF){
        i++;
    }
    fclose(f);
}

void readImage(float * data){
    FILE *fptr;
    unsigned int magic, nbImg, nbRows, nbCols;
    unsigned char val;

    //Open File
    if((fptr = fopen("train-images.idx3-ubyte","rb")) == NULL){
        printf("Can't open file");
        exit(1);
    }

    //Read File
    fread(&magic, sizeof(int), 1, fptr);
    fread(&nbImg, sizeof(int), 1, fptr);
    fread(&nbRows, sizeof(int), 1, fptr);
    fread(&nbCols, sizeof(int), 1, fptr);

    for(int i=2; i<WIDTH+2; i++){
        for(int j=2; j<HEIGHT+2; j++){ 
            fread(&val, sizeof(unsigned char), 1, fptr);  
            data[i*P+j]=(float)val/255;
        }
    }

    
}


void MatrixInit(float *M, int n, int p,int q){
    int i;
    for (i=0;i<n*p*q;i++){
        M[i]= (float)rand()/(RAND_MAX);
    }
}

void MatrixInit2(float *M, int n, int p,int q){
    int i;
    for (i=0;i<n*p*q;i++){
        M[i]= (float)rand()/(RAND_MAX/2)-1;
    }
}

void MatrixInit0(float *M, int n, int p,int q){
    int i;
    for (i=0;i<n*p*q;i++){
        M[i]= (float) 0;
    }
}


void MatrixInitId(float *M, int n, int p,int q){
    int i;
    for (i=0;i<n*p*q;i++){
        M[i]= (float) 0;
    }
    M[(n*p)/2]= (float) 1;
}

void MatrixPrint(float *M, int n, int p, int q){
    int i;
    for (i=0;i<n*p*q;i++){
        if(M[i]>0) printf(" ");
        printf("%1.2f ", M[i]);
        if ((i+1+p)%p==0 ){
            printf("\n");
        }
        if ((i+1)%(n*p)==0 ){
            printf("\n\n");
        }
    }
    printf("\n");
}

__device__ float cudaActivationTanh(float val) {
    // float temp = exp(2*val);
    // printf("%f\n",temp);
    // return (float) (temp-1)/(temp+1);
    return tanhf(val);
}


void ActivationSoftmax(float* input, size_t size) {
	int i;
	double m, sum, constant;

	m = -INFINITY;
	for (i = 0; i < size; ++i) {
		if (m < input[i]) {
			m = input[i];
		}
	}

	sum = 0.0;
	for (i = 0; i < size; ++i) {
		sum += exp(input[i] - m);
	}

	constant = m + log(sum);
	for (i = 0; i < size; ++i) {
		input[i] = exp(input[i] - constant);
	}

}



__global__ void cudaConv2D(float *img, float *kernels, float * out, int n, int p, int q, int k ) {
    // n,p= lignes,col img
    // q = nb kernels
    // k = dim kernel (k*k)

    int m=n-k+1; //dim img out
    //int id = (k-1)/2; //center

    int l = threadIdx.x; //->28 (dim img out)
    int c = threadIdx.y;  //->28 (dim img out)
    
    int d = blockIdx.x;  //->6  (dim nb kernel deconv)
    
    float temp=0;
    int i,j;
    //Calcul du bloc K*K
    for (int ki= 0; ki<k;ki++){
        for (int kj= 0; kj<k;kj++){
            i=l+ki;
            j=c+kj;   
            temp+=img[i*n + j] * kernels[d*k*k + ki*k + kj];
        }
    }

    out[d*m*m + l*m + c] = cudaActivationTanh(temp);//temp;//
}





__global__ void cudaConv3D(float *img, float *kernels, float * out, int n, int p, int q, int k ) {
    // n,p= lignes,col img
    // q = nb kernels
    // k = dim kernel (k*k)

    int m=n-k+1; //dim img out
    //int id = (k-1)/2; //center

    int l = threadIdx.x; //(dim img out)
    int c = threadIdx.y;  //(dim img out)
    
    int d = blockIdx.x;  //(dim nb kernel deconv)
    //int o = blockIdx.y; //dim nb images in

    // printf("/ %d",d);
    
    
    int i,j;
    for (int o=0; o<1; o++){
        //Calcul du bloc K*K =5*5
        float temp=0;
        for (int ki= 0; ki<k;ki++){
            for (int kj= 0; kj<k;kj++){
                i=l+ki;
                j=c+kj;   
                temp+=img[o*n*p +i*n + j] * kernels[d*k*k + ki*k + kj];
            }
        }

        //printf("/%d",o);//*m*m*16 +d*m*m+ l*m + c) ;
        // o=0-5, d=0-15, l=0-9, c=0-9
        out[ o*m*m*q +d*m*m+ l*m + c]= cudaActivationTanh(temp);
    }
}

__global__ void cudaCombine(float *in, float * out, float * id ) {
    int l = threadIdx.x; // (ligne)
    int c = threadIdx.y;  //(colonne)
    
    int g = blockIdx.x; // (dim out profondeur)

    float temp = 0;
    for(int i=0;i<6;i++){
        temp+=in[i*16*10*10 + g*10*10 +l*10 +c]*id[i*16+g];
    }
    out[g*10*10 + l*10 + c ]= temp;
}

__global__ void cudaMeanPool(float *in, float *out, int n, int p, int q) {
    // n,p= lignes,col in
    // q = nb kernels = profondeur

    int m=n/2; //dim out

    int l = threadIdx.x; //->28 (dim out)
    int c = threadIdx.y;  //->28 (dim out)
    int d = blockIdx.x;  //->6  (dim nb kernel)
    
    float temp=0;
    int i,j;
    //Calcul du bloc K*K
    for (int ki= 0; ki<2;ki++){
        for (int kj= 0; kj<2;kj++){
            i=l+ki;
            j=c+kj;    
            temp+=in[d*n*p + i*n + j];
        }
    }

    out[d*m*m + l*m + c]= temp/4;
}


__global__ void cudaFullyConnected(float *in, float *w, float *out, int n, int p,int q, int activation){
    // n, p= 5 dim in
    // q = 16 profondeur de in
    int l = threadIdx.x; // 120 = taille vecteur sortie
    float temp = 0;
    for (int i=0; i<n*p*q;i++){
        temp+=in[i]*w[i*n*p*q +l];
    }
    // 1 for tanh 2 for softmax
    if (activation ==1) {
        out[l]= cudaActivationTanh(temp); 
    }
    else if (activation ==2) {
        out[l] = temp;
    }

}



int main(){
    int L=(N-K+1); //dim out conv
    int M=(L/2); //dim out pool

    float *raw_data, *C1_data, *C1_kernel, *S2_data,*C3_dataTemp, *C3_data , *C3_kernel, *S4_data, *F5_data, *F6_data, *OUTPUT, *W1, *W2, *W3;

    srand(time(NULL));

    raw_data = (float*)malloc(N*P * sizeof(float));
    C1_data = (float*)malloc(Q*L*L * sizeof(float));
    C1_kernel = (float*)malloc(Q*K*K * sizeof(float));
    S2_data = (float*)malloc(Q*M*M * sizeof(float));

    C3_dataTemp = (float*)malloc(96*10*10 * sizeof(float));
    C3_data = (float*)malloc(16*10*10 * sizeof(float));
    C3_kernel = (float*)malloc(16*6*K*K * sizeof(float));
    S4_data = (float*)malloc(16*5*5 * sizeof(float));
    
    F5_data = (float*)malloc(120 * sizeof(float));
    F6_data = (float*)malloc(84 * sizeof(float));
    OUTPUT = (float*)malloc(10 * sizeof(float));
    W1 = (float*)malloc(120*16*5*5 * sizeof(float));
    W2 = (float*)malloc(84*120 * sizeof(float));
    W3 = (float*)malloc(84*10 * sizeof(float));

    float combineId[96] = {
        1,0,0,0,1,1,1,0,0,1,1,1,1,0,1,1,
        1,1,0,0,0,1,1,1,0,0,1,1,1,1,0,1,
        1,1,1,0,0,0,1,1,1,0,0,1,0,1,1,1,
        0,1,1,1,0,0,1,1,1,1,0,0,1,0,1,1,
        0,0,1,1,1,0,0,1,1,1,1,0,1,1,0,1,
        0,0,0,1,1,1,0,0,1,1,1,1,0,1,1,1
    };

    MatrixInit0(raw_data,N,P,1);
    readImage(raw_data);
    //MatrixPrint(raw_data,N,P,1);
    MatrixInit0(C1_data,L,L,Q);
    MatrixInit0(C1_kernel,K,K,Q);
    read_file("k1.h",C1_kernel);
    //MatrixPrint(C1_kernel,K,K,Q);

    MatrixInit0(S2_data,M,M,Q);

    MatrixInit0(C3_dataTemp,10,10,96);
    MatrixInit0(C3_data,10,10,16);
    MatrixInit0(C3_kernel,K,K,16*6);
    read_file("k2.h",C3_kernel);

    MatrixInit0(S4_data,5,5,16);

    MatrixInit0(F5_data,120,1,1);
    MatrixInit0(F6_data,84,1,1);
    MatrixInit0(OUTPUT,10,1,1);
    MatrixInit0(W1,400,120,1);
    read_file("w1.h",W1);
    MatrixInit0(W2,120,84,1);
    read_file("w2.h",W2);
    MatrixInit0(W3,84,10,1);
    read_file("w3.h",W3);


    //MatrixPrint(combineId,6,16,1);


    printf("IMAGE\n");
    //MatrixPrint(raw_data,N,P,1);
    printf("KERNEL\n");
    //MatrixPrint(C1_kernel,K,K,1);
    //GPU
    float *d_combine,*d_raw, *d_C1, *d_C1_kernel, *d_S2,*d_C3Temp, *d_C3, *d_C3_kernel, *d_S4, *d_F5, *d_F6, *d_OUTPUT, *d_W1, *d_W2, *d_W3;

    //CUDA ARRRAY---------------------------------------------------------------------
    hipMalloc((void**)&d_raw, sizeof(float)*N*P);
    hipMalloc((void**)&d_C1, sizeof(float)*Q*L*L);
    hipMalloc((void**)&d_C1_kernel, sizeof(float)*Q*K*K);
    hipMalloc((void**)&d_S2, sizeof(float)*Q*M*M);
    hipMalloc((void**)&d_C3Temp, sizeof(float)*96*10*10);
    hipMalloc((void**)&d_C3, sizeof(float)*16*10*10);
    hipMalloc((void**)&d_C3_kernel, sizeof(float)*16*6*K*K);
    hipMalloc((void**)&d_combine, sizeof(float)*16*6);
    hipMalloc((void**)&d_S4, sizeof(float)*16*5*5);
    hipMalloc((void**)&d_F5, sizeof(float)*120);
    hipMalloc((void**)&d_F6, sizeof(float)*84);
    hipMalloc((void**)&d_OUTPUT, sizeof(float)*10);
    hipMalloc((void**)&d_W1, sizeof(float)*120*400);
    hipMalloc((void**)&d_W2, sizeof(float)*120*84);
    hipMalloc((void**)&d_W3, sizeof(float)*84*10);

    hipMemcpy(d_raw, raw_data, sizeof(float) * N*P, hipMemcpyHostToDevice);
    hipMemcpy(d_C1, C1_data, sizeof(float) * Q*L*L, hipMemcpyHostToDevice);
    hipMemcpy(d_C1_kernel, C1_kernel, sizeof(float) * Q*K*K, hipMemcpyHostToDevice);
    hipMemcpy(d_S2, S2_data, sizeof(float) * Q*M*M, hipMemcpyHostToDevice);
    hipMemcpy(d_C3Temp, C3_dataTemp, sizeof(float) * 96*10*10, hipMemcpyHostToDevice);
    hipMemcpy(d_C3, C3_data, sizeof(float) * 16*10*10, hipMemcpyHostToDevice);
    hipMemcpy(d_C3_kernel, C3_kernel, sizeof(float) * 16*6*K*K, hipMemcpyHostToDevice);
    hipMemcpy(d_combine, combineId, sizeof(float) * 16*6, hipMemcpyHostToDevice);
    hipMemcpy(d_S4, S4_data, sizeof(float) * 16*5*5, hipMemcpyHostToDevice);
    hipMemcpy(d_F5, F5_data, sizeof(float) * 120, hipMemcpyHostToDevice);
    hipMemcpy(d_F6, F6_data, sizeof(float) * 84, hipMemcpyHostToDevice);
    hipMemcpy(d_OUTPUT, OUTPUT, sizeof(float) * 10, hipMemcpyHostToDevice);
    hipMemcpy(d_W1, W1, sizeof(float) * 120*400, hipMemcpyHostToDevice);
    hipMemcpy(d_W2, W2, sizeof(float) * 120*84, hipMemcpyHostToDevice);
    hipMemcpy(d_W3, W3, sizeof(float) * 10*84, hipMemcpyHostToDevice);



    //CONV 1---------------------------------------------------------------------
    dim3 nb_thread(L,L);//L=28
    dim3 nb_block(Q);//Q=6

    cudaConv2D<<<nb_block,nb_thread>>>(d_raw, d_C1_kernel, d_C1, N, P, Q, K );
    hipDeviceSynchronize();

    //MEAN 1---------------------------------------------------------------------
    dim3 nb_thread2(M,M);//M=14
    dim3 nb_block2(Q);//Q=6
    cudaMeanPool<<<nb_block2,nb_thread2>>>(d_C1, d_S2, L, L, Q );
    hipDeviceSynchronize();

    //CONV 2  14*14*6 -> 10*10*96 -> 10*10*16 ---------------------------------------------------------------------
    dim3 nb_thread3(10,10); //im out
    dim3 nb_block3(96); //nb kernel
    cudaConv3D<<<nb_block3,nb_thread3>>>(d_S2, d_C3_kernel, d_C3Temp, 14, 14, 96, K );


    //COMBINE ---------------------------------------------------------------------------------
    hipDeviceSynchronize();
    dim3 nb_thread4(10,10);
    dim3 nb_block4(16);
    cudaCombine<<<nb_block4,nb_thread4>>>(d_C3Temp,d_C3, d_combine);
    hipDeviceSynchronize();

    //MEAN 2---------------------------------------------------------------------
    dim3 nb_thread5(5,5);
    dim3 nb_block5(16);
    cudaMeanPool<<<nb_block5,nb_thread5>>>(d_C3, d_S4, 10, 10, 16 );
    hipDeviceSynchronize();


    //FC 1---------------------------------------------------------------------
    dim3 nb_thread6(120);
    cudaFullyConnected<<<1,nb_thread6>>>(d_S4, d_W1, d_F5,  5, 5, 16, 1 );
    hipDeviceSynchronize();

    //FC 2---------------------------------------------------------------------
    dim3 nb_thread7(84);
    cudaFullyConnected<<<1,nb_thread7>>>(d_F5, d_W2, d_F6,  120, 1, 1, 1 );
    hipDeviceSynchronize();

    //FC 3---------------------------------------------------------------------
    dim3 nb_thread8(10);
    cudaFullyConnected<<<1,nb_thread8>>>(d_F6, d_W3, d_OUTPUT,  84, 1, 1, 2 );
    hipDeviceSynchronize();



    //ARRAY COPY TO CPU --------------------------------------------------------------------------
    hipMemcpy(C1_data, d_C1, sizeof(float)* Q*L*L, hipMemcpyDeviceToHost);
    printf("C1\n");
    //MatrixPrint(C1_data,L,L,1);
    
    hipMemcpy(S2_data, d_S2, sizeof(float)* Q*M*M, hipMemcpyDeviceToHost);
    printf("MEAN\n");
    //MatrixPrint(S2_data,M,M,1); //M<L

    hipMemcpy(C3_dataTemp, d_C3Temp, sizeof(float)* 96*10*10, hipMemcpyDeviceToHost);
    hipMemcpy(C3_data, d_C3, sizeof(float)* 16*10*10, hipMemcpyDeviceToHost);
    
    printf("C3 Temp\n");
    //MatrixPrint(C3_dataTemp,10,10,16); 
    printf("C3\n");
    //MatrixPrint(C3_data,10,10,16); //M<L

    hipMemcpy(S4_data, d_S4, sizeof(float)* 16*5*5, hipMemcpyDeviceToHost);
    printf("MEAN2\n");
    //MatrixPrint(S4_data,5,5,2); 

    hipMemcpy(OUTPUT, d_OUTPUT, sizeof(float)* 10, hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();

    ActivationSoftmax(OUTPUT,10);
    printf("OUTPUT\n");
    MatrixPrint(OUTPUT,10,1,1); 
    
    indexMax(OUTPUT);

    hipFree(d_raw);
    hipFree(d_C1);
    hipFree(d_C1_kernel);

    free(raw_data);
    free(C1_data);
    free(C1_kernel);
}